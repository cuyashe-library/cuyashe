#include "hip/hip_runtime.h"
/**
 * cuYASHE
 * Copyright (C) 2015-2016 cuYASHE Authors
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "operators.h"

#define PRIMEP (uint64_t)18446744069414584321
#define PRIMITIVE_ROOT (int)7
#define W2 (uint64_t)16777216
#define W4 (uint64_t)281474976710656L
ZZ PZZ = to_ZZ(PRIMEP); 

#ifdef NTTMUL_TRANSFORM
  int CUDAFunctions::transform = NTTMUL;
#else
  int CUDAFunctions::transform = CUFFTMUL;
#endif

cuyasheint_t CUDAFunctions::wN = 0;
cuyasheint_t *CUDAFunctions::d_W = NULL;//W and WInv doesn't fit constant memory
cuyasheint_t *CUDAFunctions::d_WInv = NULL;
cuyasheint_t *CUDAFunctions::d_inner_results = NULL;
cuyasheint_t *CUDAFunctions::d_inner_results_used = NULL;
cuyasheint_t *CUDAFunctions::d_mulA = NULL;
cuyasheint_t *CUDAFunctions::d_mulB = NULL;
cuyasheint_t *CUDAFunctions::d_mulAux = NULL;
Complex *CUDAFunctions::d_mulComplexA = NULL;
Complex *CUDAFunctions::d_mulComplexB = NULL;
Complex *CUDAFunctions::d_mulComplexC = NULL;

/////////////
// Symbols //
/////////////
extern __constant__ cuyasheint_t M[STD_BNT_WORDS_ALLOC];
extern __constant__ int M_used;
extern __constant__ cuyasheint_t u[STD_BNT_WORDS_ALLOC];
extern __constant__ int u_used;

extern __constant__ cuyasheint_t Mpis[STD_BNT_WORDS_ALLOC*COPRIMES_BUCKET_SIZE];
extern __constant__ int Mpis_used[COPRIMES_BUCKET_SIZE];
extern __constant__ cuyasheint_t invMpis[COPRIMES_BUCKET_SIZE];

__constant__ cuyasheint_t W16[225]; 
__constant__ cuyasheint_t WInv16[225]; 
__constant__ cuyasheint_t W8[50]; 
__constant__ cuyasheint_t WInv8[50]; 

/////////////

// #elif defined(CUFFTMUL)
hipfftHandle CUDAFunctions::plan;
// #endif
int CUDAFunctions::N = 0;

__host__ __device__ inline  uint64_t s_add(uint64_t a,uint64_t b);
__host__ __device__ inline uint64_t s_sub(uint64_t a,uint64_t b);

static __device__ inline Complex ComplexMul(Complex a, Complex b);
static __device__ inline Complex ComplexAdd(Complex a, Complex b);
static __device__ inline Complex ComplexSub(Complex a, Complex b);
extern __device__ void mersenneDiv(  bn_t *x, bn_t *q, int q_bits);

///////////////////////////////////////
/// ADD
///////////////////////////////////////

#ifdef NTTMUL_TRANSFORM
__global__ void polynomialAddSub(const int OP,const cuyasheint_t *a,const cuyasheint_t *b,cuyasheint_t *c,const int size,const int N){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      if(OP == ADD){
        c[tid] = s_add(a[tid],b[tid]);
        // if(c[tid] < a[tid])
          // printf("Overflow!");
      }else
        c[tid] = s_sub(a[tid],b[tid]);
  }
}

__host__ void CUDAFunctions::callPolynomialAddSub(cuyasheint_t *c,cuyasheint_t *a,cuyasheint_t *b,int size,int OP,hipStream_t stream){
  // This method expects that both arrays are aligned
  int nthreads = 64;
  int ADDGRIDXDIM = (size%nthreads == 0? size/nthreads : size/nthreads + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(nthreads);

  polynomialAddSub <<< gridDim,blockDim,0,stream  >>> (OP,a,b,c,size,N);
  assert(hipGetLastError() == hipSuccess);
  #ifdef VERBOSE
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
}

#else
__global__ void polynomialcuFFTAddSub(const int OP,const Complex *a,const Complex *b,Complex *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      if(OP == ADD)
        c[tid] = ComplexAdd(a[tid],b[tid]);
      else
        c[tid] = ComplexSub(a[tid],b[tid]);
  }
}

__host__ void CUDAFunctions::callPolynomialcuFFTAddSub(Complex *c,Complex *a,Complex *b,int size,int OP,hipStream_t stream){
  // This method expects that both arrays are aligned
  int nthreads = 64;
  int ADDGRIDXDIM = (size%nthreads == 0? size/nthreads : size/nthreads + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(nthreads);

  polynomialcuFFTAddSub <<< gridDim,blockDim,0,stream  >>> (OP,a,b,c,size);
  assert(hipGetLastError() == hipSuccess);
  #ifdef VERBOSE
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
}

#endif

///////////////////////////////////////

///////////////////////////////////////
/// MUL

// #if defined(CUFFTMUL)

__global__ void copyIntegerToComplex(Complex *a,cuyasheint_t *b,int size){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      a[tid].x =   __ull2double_rn(b[tid]);
      // printf("%ld => %f\n\n",b[tid],a[tid].x);
      a[tid].y = 0;
  }else{
    a[tid].x = 0;
    a[tid].y = 0;
  }
}


__global__ void copyAndNormalizeComplexRealPartToInteger(cuyasheint_t *b,const Complex *a,const int size,const int N){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid < size ){
      double scale = 1.0/N;
      b[tid] = rint(a[tid].x*scale);
  }
}
////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Addition
static __device__ inline Complex ComplexAdd(Complex a, Complex b)
{
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}


static __device__ inline Complex ComplexSub(Complex a, Complex b)
{
    Complex c;
    c.x = a.x - b.x;
    c.y = a.y - b.y;
    return c;
}

// Complex multiplication
static __device__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}


// Complex pointwise multiplication
__global__ void polynomialcuFFTMul(Complex *c, const Complex *a,const Complex *b,int size){
    const int tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid < size  ){
        c[tid] = ComplexMul(a[tid], b[tid]);
    }else{
      c[tid].x = 0;
      c[tid].y = 0;
    }
}
// #elif defined(NTTMUL)

__device__ bool overflow(const uint64_t a, const uint64_t b){
  // True if a+b will result in a integer overflow.
  return (a+b) < a;
  // return lessThan((a+b),a);
}

__device__ uint64_t s_rem (uint64_t a)
{
  uint64_t res;
	
  res = a;

  if(res >= PRIMEP){
    res -= PRIMEP;
  }

  return res;
}

__device__  inline uint64_t s_mul(uint64_t a,uint64_t b){
  // Multiply and reduce a and b by prime 2^64-2^32+1
  #ifdef __CUDA_ARCH__
  const uint64_t GAP = (UINT64_MAX-PRIMEP+1);

  const uint64_t cHi = __umul64hi(a,b);
  const uint64_t cLo = a*b;


  // Reduce
  const uint64_t x3 = (cHi >> 32);
  const uint64_t x2 = (cHi & UINT32_MAX);
  const uint64_t x1 = (cLo >> 32);
  const uint64_t x0 = (cLo & UINT32_MAX);

  const uint64_t X1 = (x1<<32);
  const uint64_t X2 = (x2<<32);

  ///////////////////////////////
  //
  // Here we can see three kinds of overflows:
  //
  // * Negative overflow: Result is negative. 
  // Since uint64_t uses mod UINT64_MAX, we need to translate to the correct value mod PRIMEP.
  // * Simple overflow: Result is bigger than PRIMEP but not enough to exceed UINT64_MAX.
  //  We solve this in the same way we solve negative overflow, just translate to the correct value mod PRIMEP.
  // * Double overflow

  uint64_t res = X1+X2+x0-x2-x3;
  const bool testA = (x2+x3 > X1+X2+x0) && !( overflow(X1,X2) ||  overflow(X1+X2,x0) ); // Negative overflow
  const bool testB = ( res >= PRIMEP ); // Simple overflow
  const bool testC = (overflow(X1,X2) || overflow(X1+X2,x0)) && (X1+X2+x0 > x2+x3); // Double overflow

  // This avoids conditional branchs
  // res = (PRIMEP-res)*(testA) + (res-PRIMEP)*(!testA && testB) + (PRIMEP - (UINT64_MAX-res))*(!testA && !testB && testC) + (res)*(!testA && !testB && !testC);
  res =   (PRIMEP+res)*(testA) 
        + (res-PRIMEP)*(!testA && testB) 
        + (res+GAP)*(!testA && !testB && testC) 
        + (res)*(!testA && !testB && !testC);

   #else
  uint64_t res = (((__uint128_t)a) * ((__uint128_t)b) )%PRIMEP;
  #endif
  return res;
}
__device__ inline  uint64_t s_add(uint64_t a,uint64_t b){
  // Add and reduce a and b by prime 2^64-2^32+1
  // 4294967295L == UINT64_MAX - P
  uint64_t res = a+b;
  res += (res < a)*4294967295L;

  return s_rem(res);
}

__device__ inline uint64_t s_sub(uint64_t a,uint64_t b){
  uint64_t res;
  res = (a-b) + (b > a)*PRIMEP; 
  return res;
}

template <int RADIX, int type>
__device__ void butterfly(uint64_t *v){
  printf("Nothing to do");
}

template<>
__device__ inline void butterfly<2,FORWARD>(uint64_t *v){
  ///////////////////////
  // Radix-2 Butterfly //
  ///////////////////////
  const uint64_t v0 = s_rem(v[0]);
  const uint64_t v1 = s_rem(v[1]);
  v[0] = s_add(v0,v1);
  v[1] = s_sub(v0,v1);
}

template<>
__device__ inline void butterfly<2,INVERSE>(uint64_t *v){
  ///////////////////////
  // Radix-2 Butterfly //
  ///////////////////////
  const uint64_t v0 = s_rem(v[0]);
  const uint64_t v1 = s_rem(v[1]);
  v[0] = s_add(v0,v1);
  v[1] = s_sub(v0,v1);
}

template<>
__device__ inline void butterfly<4,FORWARD>(uint64_t *v){
  ///////////////////////
  // Radix-4 Butterfly //
  ///////////////////////
  // const uint64_t v0 = (v[0]);
  // const uint64_t v1 = (v[1]);
  // const uint64_t v2 = (v[2]);
  // const uint64_t v3 = (v[3]);
  
  // // v0 + v1 + v2 + v3
  // v[0] = s_add(s_add(s_add(v0,v1),v2),v3);
  // // v0 + W4*v1 - v2 - W4*v3
  // v[1] = s_sub(s_sub(s_add(v0,s_mul(W4,v1)),v2),s_mul(W4,v3)); 
  // // v0 - v1 + v2 - v3
  // v[2] = s_sub(s_add(s_sub(v0,v1),v2),v3);
  // // v0 - W4*v1 - v2 + W4*v3
  // v[3] = s_add(s_sub(s_sub(v0,s_mul(W4,v1)),v2),s_mul(W4,v3)); 
   
  register uint64_t s[4], temp;
  s[0] = s_add(v[0], v[2]);
  s[1] = s_sub(v[0], v[2]);
  s[2] = s_add(v[1], v[3]);
  s[3] = s_sub(v[1], v[3]);
  temp = s_mul(s[3], 48);
  v[0] = s_add(s[0], s[2]);
  v[1] = s_add(s[1], temp);
  v[2] = s_sub(s[0], s[2]);
  v[3] = s_sub(s[1], temp);
  
}

template<>
__device__ inline void butterfly<4,INVERSE>(uint64_t *v){
  ///////////////////////
  // Radix-4 Butterfly //
  ///////////////////////
  // const uint64_t v0 = (v[0]);
  // const uint64_t v1 = (v[1]);
  // const uint64_t v2 = (v[2]);
  // const uint64_t v3 = (v[3]);
  
  // // v0 + v1 + v2 + v3
  // v[0] = s_add(s_add(s_add(v0,v1),v2),v3);
  // // v0 - W4*v1 - v2 + W4*v3
  // v[1] = s_add(s_sub(s_sub(v0,s_mul(W4,v1)),v2),s_mul(W4,v3)); 
  // // v0 - v1 + v2 - v3
  // v[2] = s_sub(s_add(s_sub(v0,v1),v2),v3);
  // // v0 + W4*v1 - v2 - W4*v3
  // v[3] = s_sub(s_sub(s_add(v0,s_mul(W4,v1)),v2),s_mul(W4,v3));  

  register uint64_t s[4], temp;
  s[0] = s_add(v[0], v[2]);
  s[1] = s_sub(v[0], v[2]);
  s[2] = s_add(v[1], v[3]);
  s[3] = s_sub(v[1], v[3]);
  temp = s_mul(s[3], W4);
  v[0] = s_add(s[0], s[2]);
  v[1] = s_sub(s[1], temp);
  v[2] = s_sub(s[0], s[2]);
  v[3] = s_add(s[1], temp);
}

template<>
__device__ void butterfly<8,FORWARD>(uint64_t *v){
  ///////////////////////
  // Radix-4 Butterfly //
  ///////////////////////
  
  // register uint64 s[8], temp;
  // s[0] = s_add(v[0], v[4]);
  // s[1] = s_sub(v[0], v[4]);
  // s[2] = s_add(v[2], v[6]);
  // s[3] = s_sub(v[2], v[6]);
  // s[4] = s_add(v[1], v[5]);
  // s[5] = s_sub(v[1], v[5]);
  // s[6] = s_add(v[3], v[7]);
  // s[7] = s_sub(v[3], v[7]);
  // v[0] = s_add(s[0], s[2]);
  // v[2] = s_sub(s[0], s[2]);
  // temp = s_mul(s[3], W4);
  // v[1] = s_add(s[1], temp);
  // v[3] = s_sub(s[1], temp);
  // v[4] = s_add(s[4], s[6]);
  // v[6] = s_sub(s[4], s[6]);
  // temp = s_mul(s[7], W4);
  // v[5] = s_add(s[5], temp);
  // v[7] = s_sub(s[5], temp);
  // s[0] = s_add(v[0], v[4]);
  // s[4] = s_sub(v[0], v[4]);
  // temp = s_mul(v[5], W2);
  // s[1] = s_add(v[1], temp);
  // s[5] = s_sub(v[1], temp);
  // temp = s_mul(v[6], W4);
  // s[2] = s_add(v[2], temp);
  // s[6] = s_sub(v[2], temp);
  // temp = s_mul(v[7], 72);
  // s[3] = s_add(v[3], temp);
  // s[7] = s_sub(v[3], temp);
  // v[0] = s[0];
  // v[1] = s[1];
  // v[2] = s[2];
  // v[3] = s[3];
  // v[4] = s[4];
  // v[5] = s[5];
  // v[6] = s[6];
  // v[7] = s[7];
}

// template<>
// __device__ void butterfly<8,INVERSE>(uint64_t *v){
//   ///////////////////////
//   // Radix-4 Butterfly //
//   ///////////////////////
//   const uint64_t v0 = s_rem(v[0]);
//   const uint64_t v1 = s_rem(v[1]);
//   const uint64_t v2 = s_rem(v[2]);
//   const uint64_t v3 = s_rem(v[3]);
//   const uint64_t v4 = s_rem(v[4]);
//   const uint64_t v5 = s_rem(v[5]);
//   const uint64_t v6 = s_rem(v[6]);
//   const uint64_t v7 = s_rem(v[7]);
  

//   v[0] = s_add(s_add(s_add(s_add(s_add(s_add(s_add(v0,v1),v2),v3),v4),v5),v6),v7);
//   v[1] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[1])),
//                                                     s_mul(v2,WInv8[2])),
//                                                     s_mul(v3,WInv8[3])),
//                                                     s_mul(v4,WInv8[4])),
//                                                     s_mul(v5,WInv8[5])),
//                                                     s_mul(v6,WInv8[6])),
//                                                     s_mul(v7,WInv8[7]));
//   v[2] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[2])),
//                                                     s_mul(v2,WInv8[4])),
//                                                     s_mul(v3,WInv8[6])),
//                                                     s_mul(v4,WInv8[8])),
//                                                     s_mul(v5,WInv8[10])),
//                                                     s_mul(v6,WInv8[12])),
//                                                     s_mul(v7,WInv8[14]));
//   v[3] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[3])),
//                                                     s_mul(v2,WInv8[6])),
//                                                     s_mul(v3,WInv8[9])),
//                                                     s_mul(v4,WInv8[12])),
//                                                     s_mul(v5,WInv8[15])),
//                                                     s_mul(v6,WInv8[18])),
//                                                     s_mul(v7,WInv8[21]));
//   v[4] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[4])),
//                                                     s_mul(v2,WInv8[8])),
//                                                     s_mul(v3,WInv8[12])),
//                                                     s_mul(v4,WInv8[16])),
//                                                     s_mul(v5,WInv8[20])),
//                                                     s_mul(v6,WInv8[24])),
//                                                     s_mul(v7,WInv8[28]));
//   v[5] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[5])),
//                                                     s_mul(v2,WInv8[10])),
//                                                     s_mul(v3,WInv8[15])),
//                                                     s_mul(v4,WInv8[20])),
//                                                     s_mul(v5,WInv8[25])),
//                                                     s_mul(v6,WInv8[30])),
//                                                     s_mul(v7,WInv8[35]));
//   v[6] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[6])),
//                                                     s_mul(v2,WInv8[12])),
//                                                     s_mul(v3,WInv8[18])),
//                                                     s_mul(v4,WInv8[24])),
//                                                     s_mul(v5,WInv8[30])),
//                                                     s_mul(v6,WInv8[36])),
//                                                     s_mul(v7,WInv8[42]));
//   v[7] = s_add(s_add(s_add(s_add(s_add(s_add(s_add( v0,
//                                                     s_mul(v1,WInv8[7])),
//                                                     s_mul(v2,WInv8[14])),
//                                                     s_mul(v3,WInv8[21])),
//                                                     s_mul(v4,WInv8[28])),
//                                                     s_mul(v5,WInv8[35])),
//                                                     s_mul(v6,WInv8[42])),
//                                                     s_mul(v7,WInv8[49]));
// }

__device__ int expand(int idxL, int N1, int N2){
	return (idxL/N1)*N1*N2 + (idxL%N1);
}

__global__ void NTTScale(cuyasheint_t *data,const int size,const int N){
  const unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
  // const unsigned int cid = (tid/N)*N + (tid%N); // residueId*resideSize + coefficient 
  if( tid < size )
    data[tid] /= N;
} 

template<int RADIX, int type>
__device__ void NTTIteration(cuyasheint_t *W,
                                      cuyasheint_t *WInv,
                                      const int residue_index,
                                      const int j,
                                      const int N,
                                      const int Ns,
                                      const cuyasheint_t* data0,
                                      cuyasheint_t *data1){
	uint64_t v[RADIX] = {0};
	const int idxS = j+residue_index;
  int w_index = ((j%Ns)*N)/(Ns*RADIX);
  const int idxD = expand(j,Ns,RADIX)+residue_index;
  if(idxD == 151 || idxD+Ns == 151)
    v[0] = v[0];

  for(int r=0; r<RADIX; r++)
    if(type == FORWARD)
      v[r] = s_mul(data0[idxS+r*N/RADIX],W[w_index*r]);
    else
      v[r] = s_mul(data0[idxS+r*N/RADIX],WInv[w_index*r]);

  butterfly<RADIX,type>(v);

	// const int idxD = expand(j,Ns,RADIX)+residue_index;
	for(int r=0; r<RADIX;r++)
  		data1[idxD+r*Ns] = v[r];
  
}

template<int RADIX, int type>
__global__ void NTT(cuyasheint_t *d_W,cuyasheint_t *d_WInv,const int N, const int Ns, cuyasheint_t* dataI, cuyasheint_t* dataO){

  const int residue_index = (blockIdx.x)*N;
  for(int i = 0; i < N/RADIX; i += 1024){
    // " Threads virtuais "
    const int j = (threadIdx.x+i);
    if( j < N)
      NTTIteration<RADIX,type>(d_W,d_WInv,residue_index,j, N, Ns, dataI, dataO);
    __syncthreads();
  }
}

__host__ void CUDAFunctions::callNTT(const int N, const int NPolis,int RADIX, cuyasheint_t* dataI, cuyasheint_t* dataO,const int type){

  dim3 blockDim(std::min(N/RADIX,1024));
  dim3 gridDim(NPolis);

  for(int Ns=1; Ns<N; Ns*=RADIX){
    if(RADIX == 4){
      if(type == FORWARD)
        NTT<4,FORWARD><<<gridDim,blockDim>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,dataI,dataO);
      else
        NTT<4,INVERSE><<<gridDim,blockDim>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,dataI,dataO);
    }
    else{
      assert(RADIX == 2);
      if(type == FORWARD)
        NTT<2,FORWARD><<<gridDim,blockDim>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,dataI,dataO);
      else
        NTT<2,INVERSE><<<gridDim,blockDim>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,dataI,dataO);
    }
    assert(hipGetLastError() == hipSuccess);
    std::swap(dataI,dataO);
  }
}

__global__ void polynomialNTTMul(cuyasheint_t *c, const cuyasheint_t *a,const cuyasheint_t *b,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      uint64_t a_value = a[tid];
      uint64_t b_value = b[tid];

      c[tid] = s_mul(a_value,b_value);
  }
}

__global__ void polynomialNTTAdd(cuyasheint_t *a,const cuyasheint_t *b,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      uint64_t a_value = a[tid];
      uint64_t b_value = b[tid];

      // In-place
      a[tid] = s_add(a_value,b_value);
      // a[tid] = a_value*b_value % 18446744069414584321;
  }
}
// #endif

__global__ void polynomialOPInteger(const int opcode,
                                      const cuyasheint_t *a,
                                      const cuyasheint_t integer_array,
                                      cuyasheint_t *output,
                                      const int N,
                                      const int NPolis){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int size = N*NPolis;
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int cid = tid % N; // Coefficient id
  const int rid = tid / N; // Residue id

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
    cuyasheint_t operand = integer_array;

    switch(opcode)
    {
    case ADD:
      if(cid == 0)
        output[tid] = (a[tid] + operand) % CRTPrimesConstant[rid];
      break;
    case SUB:
      if(cid == 0){
        if(a[tid] > operand)
          output[tid] = (a[tid] - operand) % CRTPrimesConstant[rid];
        else
          output[tid] = ((CRTPrimesConstant[rid]-operand) + a[tid]) % CRTPrimesConstant[rid];
      }
      break;
    case MUL:
        output[tid] = (a[tid] * operand)% CRTPrimesConstant[rid];
      break;
    default:
      //This case shouldn't be used. 
      assert(1 == 0);
      break;
    }
  }

}

// Operations between polynomials and integers
__host__ void CUDAFunctions::callPolynomialOPInteger(
                                                              const int opcode,
                                                              hipStream_t stream,
                                                              cuyasheint_t *b,
                                                              cuyasheint_t *a,
                                                              cuyasheint_t integer_array,
                                                              const int N,
                                                              const int NPolis)
{
  // This method applies a 0-degree operation over all CRT residues
  const int size = N*NPolis;

  const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  const dim3 gridDim(ADDGRIDXDIM);
  const dim3 blockDim(ADDBLOCKXDIM);

  polynomialOPInteger<<< gridDim,blockDim, 0, stream>>> ( opcode,
                                                          a,
                                                          integer_array,
                                                          b,
                                                          N,
                                                          NPolis);
  assert(hipGetLastError() == hipSuccess);

}

__global__ void polynomialcuFFTOPInteger( const int opcode,
                                          const Complex *a,
                                          const cuyasheint_t integer,
                                          Complex *output,
                                          const int N,
                                          const int NPolis){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int size = N*NPolis;
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int cid = tid % N; // Coefficient id
  // const int rid = tid / N; // Residue id

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
    Complex operand = {__ull2double_rn(integer),0};

    switch(opcode)
    {
    case ADD:
      if(cid == 0)
        output[tid] = ComplexAdd(a[tid],operand) ;
      break;
    case SUB:
      if(cid == 0)
        output[tid] = ComplexSub(a[tid],operand);
      break;
    case MUL:
        output[tid] = ComplexMul(a[tid] , operand);
      break;
    default:
      //This case shouldn't be used. 
      assert(1 == 0);
      break;
    }
  }

}

__host__ void CUDAFunctions::callPolynomialcuFFTOPInteger(
                                                              const int opcode,
                                                              hipStream_t stream,
                                                              Complex *b,
                                                              Complex *a,
                                                              cuyasheint_t integer,
                                                              const int N,
                                                              const int NPolis)
{
  // This method applies a 0-degree operation over all CRT residues
  const int size = N*NPolis;

  const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  const dim3 gridDim(ADDGRIDXDIM);
  const dim3 blockDim(ADDBLOCKXDIM);

  // This function must be refactored. It works for multiplications but not 
  // for small additions. In that case the input integer must be converted to a 
  // 0-degree polynomial.
  if(opcode == ADD){
    log_error("Do not use polynomialcuFFTOPInteger function for additions.");
    exit(1);
  }

  polynomialcuFFTOPInteger<<< gridDim,blockDim, 0, stream>>> ( opcode,
                                                                a,
                                                                integer,
                                                                b,
                                                                N,
                                                                NPolis);
  assert(hipGetLastError() == hipSuccess);

}

__global__ void polynomialOPDigit(const int opcode,
                                      bn_t *b,
                                      bn_t *a,
                                      const bn_t digit,
                                      const int N
                                  ){

  // We have one thread per polynomial coefficient on 32 threads-block.
  const int size = N;
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  int nwords = 0;
  cuyasheint_t carry = 0;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
    switch(opcode)
    {
    case ADD:
      if(tid == 0){

        nwords = max_d(a[tid].used,digit.used);
        carry = bn_addn_low(b[tid].dp, a[tid].dp, digit.dp,nwords);
        b[tid].used = nwords;

        /* Equivalent to "If has a carry, add as last word" */
        b[tid].dp[b[tid].used] = carry;
        b[tid].used += (carry > 0);
      }
      break;
    case MUL:
      assert(a[tid].alloc >= STD_BNT_WORDS_ALLOC);
      assert(digit.alloc >= STD_BNT_WORDS_ALLOC);
      bn_zero_non_used(&b[tid]);
      bn_muln_low(b[tid].dp,
                  a[tid].dp,
                  digit.dp,
                  STD_BNT_WORDS_ALLOC);
      b[tid].used = a[tid].used + 1;
      bn_adjust_used(&b[tid]);
      break;
    default:
      //This case shouldn't be used. 
      assert(1 == 0);
      break;
    }
  }

}

__host__ void CUDAFunctions::callPolynomialOPDigit( const int opcode,
                                                            hipStream_t stream,
                                                            bn_t *b,
                                                            bn_t *a,
                                                            bn_t digit,
                                                            const int N){
    // This method applies a 0-degree operation over all coeficients
  const int size = N;

  const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  const dim3 gridDim(ADDGRIDXDIM);
  const dim3 blockDim(ADDBLOCKXDIM);

  polynomialOPDigit<<< gridDim,blockDim, 1, stream>>> ( opcode,
                                                        b,
                                                        a,
                                                        digit,
                                                        N);
  assert(hipGetLastError() == hipSuccess);
  return;
}

__host__ cuyasheint_t* CUDAFunctions::applyNTT( cuyasheint_t *d_a,
                                                const int N,
                                                const int NPolis,
                                                int type,
                                                hipStream_t stream){
  if(N != CUDAFunctions::N)
    CUDAFunctions::init(N/2);

  hipError_t result;
  const int size = N*NPolis;
  // cuyasheint_t *aux = CUDAFunctions::d_mulAux;
  cuyasheint_t *aux;
  result = hipMalloc((void**)&aux,size*sizeof(cuyasheint_t));

  result = hipMemsetAsync(aux,0,size*sizeof(cuyasheint_t),stream);
  assert(result == hipSuccess);

  int RADIX;
  /*if(N % 8 == 0)
    RADIX = 8;
  else*/
  if(is_power_of(N,4))
    RADIX = 4;
  else{
    assert(is_power_of(N,2));
    RADIX = 2;
    }
  dim3 blockDim(std::min(N/RADIX,1024));
  dim3 gridDim(NPolis);

  // Forward
  for(int Ns=1; Ns<N; Ns*=RADIX){
    if(type == FORWARD){ 
      /*if(RADIX == 8)
        NTT<8,FORWARD><<<gridDim,blockDim,1,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      else*/ if(RADIX == 4)
        NTT<4,FORWARD><<<gridDim,blockDim,0,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      else{
        assert(RADIX == 2);
        NTT<2,FORWARD><<<gridDim,blockDim,0,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      }
    }else{      
      /*if(RADIX == 8)
        NTT<8,FORWARD><<<gridDim,blockDim,0,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      else*/ if(RADIX == 4)
        NTT<4,INVERSE><<<gridDim,blockDim,0,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      else{
        assert(RADIX == 2);
        NTT<2,INVERSE><<<gridDim,blockDim,0,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,Ns,d_a,aux);
      }
    }
    assert(hipGetLastError() == hipSuccess);
    std::swap(aux,d_a);
  }
  if(type == INVERSE){
    // std::swap(aux,d_a);
    dim3 blockDimMul(ADDBLOCKXDIM);
    dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
    NTTScale<<< gridDimMul,blockDimMul,0,stream >>>(d_a,size,N);
    assert(hipGetLastError() == hipSuccess);
  }
  return d_a;
}

__host__ void CUDAFunctions::executeCopyIntegerToComplex(   Complex *d_a, 
                                                            cuyasheint_t *a,
                                                            const int size,
                                                            hipStream_t stream){
  dim3 blockDim(32);
  dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

  copyIntegerToComplex<<< gridDim,blockDim,0,stream >>>(d_a,a,size);

  assert(hipGetLastError() == hipSuccess);
}

__host__ void CUDAFunctions::executeCopyAndNormalizeComplexRealPartToInteger(   cuyasheint_t *d_a, 
                                                                                hipfftDoubleComplex *a,
                                                                                const int size,
                                                                                int N,
                                                                                hipStream_t stream){
  dim3 blockDim(32);
  dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

  copyAndNormalizeComplexRealPartToInteger<<< gridDim,blockDim,0,stream >>>( d_a,
                                                                            a,
                                                                            size,
                                                                            N);

  assert(hipGetLastError() == hipSuccess);
}

__host__ void CUDAFunctions::executeNTTScale(   cuyasheint_t *d_result, 
                                                const int size, 
                                                const int N,
                                                hipStream_t stream){
  dim3 blockDimMul(ADDBLOCKXDIM);
  dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
  NTTScale<<< gridDimMul,blockDimMul,0,stream >>>(d_result,size,N);
  assert(hipGetLastError() == hipSuccess);
}

__host__ void CUDAFunctions::executeCuFFTPolynomialMul( Complex *c, 
                                                        Complex *a, 
                                                        Complex *b, 
                                                        int size, 
                                                        hipStream_t stream){
  dim3 blockDim(32);
  dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

  polynomialcuFFTMul<<<gridDim,blockDim,0,stream>>>(c,a,b,size);

  assert(hipGetLastError() == hipSuccess);
}
__host__ void CUDAFunctions::executePolynomialMul(cuyasheint_t *c, 
                                                  cuyasheint_t *a, 
                                                  cuyasheint_t *b, 
                                                  const int size, 
                                                  hipStream_t stream){
  dim3 blockDimMul(ADDBLOCKXDIM);
  dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
  polynomialNTTMul<<<gridDimMul,blockDimMul,0,stream>>>(c,a,b,size);
  assert(hipGetLastError() == hipSuccess);
}

__host__ void CUDAFunctions::executePolynomialAdd(cuyasheint_t *c, 
                                                  cuyasheint_t *a, 
                                                  cuyasheint_t *b, 
                                                  const int size, 
                                                  hipStream_t stream){
  dim3 blockDimMul(ADDBLOCKXDIM);
  dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
  polynomialNTTAdd<<<gridDimMul,blockDimMul,0,stream>>>(a,b,size);
  assert(hipGetLastError() == hipSuccess);
}

/**
 * Returns true if a is power of b
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__host__ bool is_power_of(uint64_t a, uint64_t b){
  assert(b > 1);
  
  uint64_t n = a;
  while (n % b == 0)
    n /= b;
  return (n==1);
}



__host__ cuyasheint_t* CUDAFunctions::callPolynomialMul(cuyasheint_t *output,
                                                        cuyasheint_t *a,
                                                        cuyasheint_t *b,
                                                        const int size,
                                                        hipStream_t stream){
  // This method expects that both arrays are aligned

  // Input:
  // stream: cudaStream
  // a: first operand
  // realign_A: flag. true if this operand need to be realign
  // A_N: number of coefficients for each operand. Used only if we need to realign this
  // b: second operand
  // realign_B: flag. true if this operand need to be realign
  // B_N: number of coefficients for each residue. Used only if we need to realign this
  // N: number of coefficients for each residue. This is the N that should be considered for the operation.
  // NPolis: number of residues
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  assert(N == CUDAFunctions::N);
  cuyasheint_t *d_result = output;
  // hipError_t result;

  // #ifdef NTTMUL
  if(transform == NTTMUL){

    // Multiply
    dim3 blockDimMul(ADDBLOCKXDIM);
    dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
    polynomialNTTMul<<<gridDimMul,blockDimMul,0,stream>>>(output,a,b,size);
    assert(hipGetLastError() == hipSuccess);

    // result = hipDeviceSynchronize();
  }else{
  
    /////////////
    // cuFFT  //
    ///////////
    // dim3 blockDim(32);
    // dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

    // polynomialcuFFTMul<<<gridDim,blockDim,0,stream>>>(d_a,d_b,d_c,size);
    // assert(hipGetLastError() == hipSuccess);
  }

  return d_result;
}

/**
 * [CUDAFunctions::init description]
 * @param N The target polynomial degree
 */
__host__ void CUDAFunctions::init(int M){
  int N = 2*M;
  CUDAFunctions::N = N;
  hipError_t result;

  // #ifdef NTTMUL
  std::cout << "CUDAFunctions initializing  = " << N << std::endl;

  cuyasheint_t *h_W;
  cuyasheint_t *h_WInv;

  assert((PZZ-1)%(N) == 0);

  //////////////
  // Builds wN //
  //////////////
  cuyasheint_t k = conv<cuyasheint_t>(PZZ-1)/N;
  ZZ wNZZ = NTL::PowerMod(ZZ(PRIMITIVE_ROOT),k,PZZ);

  wN = conv<cuyasheint_t>(wNZZ);
  h_W = (cuyasheint_t*)malloc(N*sizeof(cuyasheint_t));
  result = hipMalloc((void**)&d_W,N*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  h_WInv = (cuyasheint_t*)malloc(N*sizeof(cuyasheint_t));
  result = hipMalloc((void**)&d_WInv,N*sizeof(cuyasheint_t));
  assert(result == hipSuccess);

  // Computes 1-th column from W
  for(int j = 0; j < N; j++)
    h_W[j] = conv<cuyasheint_t>(NTL::PowerMod(wNZZ,j,PZZ));
  

  // Computes 1-th column from WInv
  for(int j = 0; j < N; j++)
      h_WInv[j] = conv<cuyasheint_t>(NTL::InvMod(conv<ZZ>(h_W[j]),PZZ ));

  result = hipMemcpy (d_W,h_W , N*sizeof(cuyasheint_t),hipMemcpyHostToDevice);
  assert(result == hipSuccess);
  result = hipMemcpy(d_WInv,h_WInv , N*sizeof(cuyasheint_t),hipMemcpyHostToDevice);
  assert(result == hipSuccess);

  free(h_W);
  free(h_WInv);

  // int RADIX_N = 8;
  // // W used on radix-8 NTT
  // #ifdef VERBOSE
  // std::cout << "Will compute W -- N = 8" << std::endl;
  // std::cout << "P " << PZZ << std::endl;
  // #endif

  // k = conv<cuyasheint_t>(PZZ-1)/RADIX_N;
  // wNZZ = NTL::PowerMod(ZZ(PRIMITIVE_ROOT),k,PZZ);

  // wN = conv<cuyasheint_t>(wNZZ);
  
  // int VALUES_TO_COMPUTE = (RADIX_N - 1)*(RADIX_N - 1)+1;
  // h_W = (cuyasheint_t*)malloc(VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // h_WInv = (cuyasheint_t*)malloc(VALUES_TO_COMPUTE*sizeof(cuyasheint_t));

  // // Computes 1-th column from W
  // for(int j = 0; j < VALUES_TO_COMPUTE; j++)
  //   h_W[j] = conv<cuyasheint_t>(NTL::PowerMod(wNZZ,j,PZZ));

  // // Computes 1-th column from WInv
  // for(int j = 0; j < VALUES_TO_COMPUTE; j++)
  //     h_WInv[j] = conv<cuyasheint_t>(NTL::InvMod(conv<ZZ>(h_W[j]),PZZ ));

  // result = hipMemcpyToSymbol(HIP_SYMBOL(W8),h_W, VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // assert(result == hipSuccess);
  // result = hipMemcpyToSymbol(HIP_SYMBOL(WInv8),h_WInv, VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // assert(result == hipSuccess);

  // free(h_W);
  // free(h_WInv);

  // RADIX_N = 16;
  // // W used on radix-16 NTT
  // #ifdef VERBOSE
  // std::cout << "Will compute W -- N = 16" << std::endl;
  // std::cout << "P " << PZZ << std::endl;
  // #endif

  // k = conv<cuyasheint_t>(PZZ-1)/RADIX_N;
  // wNZZ = NTL::PowerMod(ZZ(PRIMITIVE_ROOT),k,PZZ);

  // wN = conv<cuyasheint_t>(wNZZ);
  
  // VALUES_TO_COMPUTE = (RADIX_N - 1)*(RADIX_N - 1)+1;
  // h_W = (cuyasheint_t*)malloc(VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // h_WInv = (cuyasheint_t*)malloc(VALUES_TO_COMPUTE*sizeof(cuyasheint_t));

  // // Computes 1-th column from W
  // for(int j = 0; j < VALUES_TO_COMPUTE; j++)
  //   h_W[j] = conv<cuyasheint_t>(NTL::PowerMod(wNZZ,j,PZZ));

  // // Computes 1-th column from WInv
  // for(int j = 0; j < VALUES_TO_COMPUTE; j++)
  //     h_WInv[j] = conv<cuyasheint_t>(NTL::InvMod(conv<ZZ>(h_W[j]),PZZ ));

  // result = hipMemcpyToSymbol(HIP_SYMBOL(W8),h_W, VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // assert(result == hipSuccess);
  // result = hipMemcpyToSymbol(HIP_SYMBOL(WInv8),h_WInv, VALUES_TO_COMPUTE*sizeof(cuyasheint_t));
  // assert(result == hipSuccess);

  // free(h_W);
  // free(h_WInv);

  hipfftResult fftResult;
    // # of CRT residues
  const int batch = CRTPrimes.size();
  assert(batch > 0);

  // # 1 dimensional FFT
  const int rank = 1;

  // No idea what is this
  int n[1] = {N};


  fftResult = hipfftPlanMany(&CUDAFunctions::plan, rank, n,
       NULL, 1, N,  //advanced data layout, NULL shuts it off
       NULL, 1, N,  //advanced data layout, NULL shuts it off
       HIPFFT_Z2Z, batch);
  // fftResult = hipfftPlan1d(&CUDAFunctions::plan, N, HIPFFT_Z2Z, 1);


  assert(fftResult == HIPFFT_SUCCESS);
  std::cout << "Plan created with signal size " << N << std::endl;
  /**
   * Alloc memory for d_inner_results
   */
  const unsigned int size = N*CRTPrimes.size();

  result = hipMalloc((void**)&CUDAFunctions::d_inner_results, size*STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&CUDAFunctions::d_inner_results_used, size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);


  /**
   * Pre-allocated arrays for NTT multiplication
   */
    
  result = hipMalloc((void**)&CUDAFunctions::d_mulA,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&CUDAFunctions::d_mulB,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&CUDAFunctions::d_mulAux,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);


  /**
   * Pre-allocated arrays for FFT multiplication
   */
    
  result = hipMalloc((void**)&CUDAFunctions::d_mulComplexA,size*sizeof(Complex));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&CUDAFunctions::d_mulComplexB,size*sizeof(Complex));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&CUDAFunctions::d_mulComplexC,size*sizeof(Complex));
  assert(result == hipSuccess);
}

__global__ void cuICRTFix(bn_t *a, const int N, bn_t q,bn_t u_q,bn_t q2){
  //////////////////////////////////////////////////////
  // This kernel must be executed with N threads //
  //////////////////////////////////////////////////////
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;

  if(tid < N){
    bn_t coef = a[tid];
    coef.used = get_used_index(coef.dp,STD_BNT_WORDS_ALLOC)+1;

    if(bn_cmp_abs(&coef,&q2) == CMP_GT){
      /**
       * If coef > q^2, it is result of an underflow on polynomial reduction 
       * during the polynomial reduction.
       *
       * coef = q - (M - coef)
       */
      // result = M - coef
      int carry = bn_subn_low(coef.dp, M, coef.dp, max_d(coef.used,M_used));
      assert(carry == BN_POS);     
      coef.used = get_used_index(coef.dp,STD_BNT_WORDS_ALLOC)+1;
      // result = q - result
      carry = bn_subn_low(coef.dp,q.dp,coef.dp,max_d(coef.used,q.used));
      coef.used = get_used_index(coef.dp,STD_BNT_WORDS_ALLOC)+1;
    }  
    // result = result % q
    bn_mod_barrt( &coef,
                  coef,
                  q.dp,
                  q.used,
                  u_q.dp,
                  u_q.used); 
    a[tid] = coef;
    bn_zero_non_used(&a[tid]);
  }
}

// __global__ void polynomialReductionCRT( cuyasheint_t *a,
//                                         const int half,
//                                         const int N,
//                                         const int NPolis){     
//   // This kernel must have (N-half)*Npolis threads

//   const int tid = threadIdx.x + blockIdx.x*blockDim.x;
//   const int rid = tid / (N-half); 
//   const int cid = tid % (N-half);

//   if( (cid+half+1 < N) && (rid*N + cid + half + 1 < N*NPolis)){
//     // assert(a[rid*N + cid] < CRTPrimesConstant[rid]);
//     // assert(a[rid*N + cid + half + 1] < CRTPrimesConstant[rid]);
//     a[rid*N + cid] %= CRTPrimesConstant[rid];
//     a[rid*N + cid + half + 1] %= CRTPrimesConstant[rid];

//     // bool is_neg = (a[rid*N + cid] < a[rid*N + cid + half + 1]);
//     a[rid*N + cid] -= a[rid*N + cid + half + 1];
//     // a[rid*N + cid] += is_neg*CRTPrimesConstant[rid]*CRTPrimesConstant[rid];
//     __syncthreads();
//     a[rid*N + cid + half + 1] = 0;
//     a[rid*N + cid] %= CRTPrimesConstant[rid];
//   }

// }

__global__ void polynomialReductionCoefs( bn_t *a,
                                          const int half,
                                          const int N ){     
  ////////////////////////////////////////////////////////
  // This kernel must be executed with (N-half) threads //
  ////////////////////////////////////////////////////////

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int cid = tid % (N-half);

  if(tid < (N-half) && cid + half + 1 < N && a[cid+half+1].used > 0){
    bn_adjust_used(&a[cid]);
    bn_adjust_used(&a[cid+half+1]);
    /////////////
    // a % phi //
    /////////////
    // a[i] = a[i] - a[i+half]
    int carry = bn_subn_low(  a[cid].dp,
                              a[cid].dp,
                              a[cid + half + 1].dp,
                              max_d(a[cid].used, a[cid + half + 1].used)
                            );
    a[cid].used = max_d(a[cid].used, a[cid + half + 1].used);
    bn_adjust_used(&a[cid]);
    a[cid].sign = carry;

    if(carry == BN_NEG){
      // two's complement of a's words
      // two's complement of x is equal to the complement of x plus 1
      a[cid].dp[0] = (~a[cid].dp[0]) + 1;
      for(int i = 1; i < a[cid].used; i++)
        a[cid].dp[i] = (~a[cid].dp[i]);
    }
    //   // (a-b) % q
    //   mersenneDiv(&a[cid], &q, q_bits);           
    //   bn_adjust_used(&a[cid]);

    //   // // q - ((a-b) % q)
    //   carry = bn_subn_low(  a[cid].dp,
    //                         q.dp,
    //                         a[cid].dp,
    //                         q.used );

    //   assert(carry == BN_POS);
    //   a[cid].used = q.used;
    //   bn_adjust_used(&a[cid]);
    // }
    bn_zero(&a[cid + half + 1]);
  }

}

__host__ void CUDAFunctions::callPolynomialReductionCoefs(  bn_t *a,
                                                            const int half,
                                                            const int N ){  
    const int size = (N-half);

    dim3 blockDim(ADDBLOCKXDIM);
    dim3 gridDim(size/ADDBLOCKXDIM + (size % ADDBLOCKXDIM == 0? 0:1));
    /**
     * Polynomial reduction
     */
    polynomialReductionCoefs<<< gridDim,blockDim, 0, NULL>>>( a,
                                                              half,
                                                              N);
    hipError_t result = hipGetLastError();
    assert(result == hipSuccess);

}

__host__ void  CUDAFunctions::write_crt_primes(){

  #ifdef VERBOSE
  std::cout << "primes: "<< std::endl;
  for(unsigned int i = 0; i < CRTPrimes.size();i++)
    std::cout << CRTPrimes[i] << " ";
  std::cout << std::endl;
  #endif
  
  // Choose what memory will be used to story CRT Primes
  if(CRTPrimes.size() < MAX_PRIMES_ON_C_MEMORY){
    
    #ifdef VERBOSE
    std::cout << "Writting CRT Primes to GPU's constant memory" << std::endl;
    #endif

    /////////////////
    // Copy primes //
    /////////////////
    hipError_t result = hipMemcpyToSymbol(HIP_SYMBOL( CRTPrimesConstant),
                                                    &(CRTPrimes[0]),
                                                    CRTPrimes.size()*sizeof(cuyasheint_t),
                                                    0,
                                                   hipMemcpyHostToDevice
                                            );
    assert(result == hipSuccess);

    ////////////
    // Copy M //
    ////////////
    
    bn_t h_M;
    
    get_words_host(&h_M,CRTProduct);
    assert(h_M.alloc >= STD_BNT_WORDS_ALLOC);
    result = hipMemcpyToSymbol(HIP_SYMBOL(M),h_M.dp, h_M.used*sizeof(cuyasheint_t),0,hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    result = hipMemcpyToSymbol(HIP_SYMBOL(M_used),&h_M.used, sizeof(int),0,hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    
    ////////////
    // Copy u //
    ////////////

    cuyasheint_t *h_u;
    bn_t d_u = get_reciprocal(CRTProduct);
    h_u = (cuyasheint_t*)malloc(d_u.alloc*sizeof(cuyasheint_t));
    // assert(d_u.alloc >= STD_BNT_WORDS_ALLOC);
    result = hipMemcpy(h_u,d_u.dp,d_u.alloc*sizeof(cuyasheint_t),hipMemcpyDeviceToHost);
    assert(result == hipSuccess);
    result = hipMemcpyToSymbol(HIP_SYMBOL(u),h_u,d_u.alloc*sizeof(cuyasheint_t),0,hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    result = hipMemcpyToSymbol(HIP_SYMBOL(u_used),&d_u.used, sizeof(int),0,hipMemcpyHostToDevice);
    assert(result == hipSuccess);
    
    //////////////
    // Copy Mpi //
    //////////////    
    bn_t *h_Mpis;
    h_Mpis = (bn_t*) malloc( CRTPrimes.size()*sizeof(bn_t) );
    
    for(unsigned int i = 0; i < CRTPrimes.size();i++){
      h_Mpis[i].alloc = 0;
      get_words_host(&h_Mpis[i],CRTMpi[i]);
      result = hipMemcpyToSymbol(HIP_SYMBOL(Mpis), h_Mpis[i].dp, STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t),i*STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t),hipMemcpyHostToDevice);
      assert(result == hipSuccess);
      result = hipMemcpyToSymbol(HIP_SYMBOL(Mpis_used),&h_Mpis[i].used, sizeof(int),i*sizeof(int),hipMemcpyHostToDevice);
      assert(result == hipSuccess);
    }

    /////////////////
    // Copy InvMpi //
    /////////////////

  result = hipMemcpyToSymbol(HIP_SYMBOL(invMpis),
                &CRTInvMpi[0],
                CRTPrimes.size()*sizeof(cuyasheint_t),
                0,
                hipMemcpyHostToDevice
              );
    assert(result == hipSuccess);

    ////////////////////
    // Release memory //
    ////////////////////
    result = hipDeviceSynchronize();
    assert(result == hipSuccess);
    for(unsigned int i = 0; i < CRTPrimes.size();i++)
      free(h_Mpis[i].dp);

    free(h_Mpis);
    free(h_M.dp);
    hipFree(d_u.dp);
  }else{
    throw "Too many primes.";
  }
}
